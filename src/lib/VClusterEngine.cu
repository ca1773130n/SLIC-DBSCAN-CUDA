#include "hip/hip_runtime.h"
#include <sdbscan/VClusterEngine.h>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

namespace sdbscan {
	__global__ void breadthFirstSearch_device(SegNode *N, int numV, int *E, unsigned char *Fa, unsigned char *Xa);
	__global__ void makeGraph_device_step1(SPixelInfo* si, SegNode *nodes, int *Va1, int numV, float Rs, float Rc);
	__global__ void makeGraph_device_step2(SPixelInfo* si, SegNode *nodes, int *Va1, int *Va2, int *e, int numV, float Rs, float Rc);

	__device__ float distanceS(SPixelInfo *si1, SPixelInfo *si2) {
		float diffX = fabs(si1->center.x - si2->center.x);
		float diffY = fabs(si1->center.y - si2->center.y);
		float posDiff = sqrtf(diffX * diffX + diffY * diffY);
		return posDiff;
	}

	__device__ float distanceC(SPixelInfo *si1, SPixelInfo *si2) {
		float diffR = fabs(si1->color_info.r - si2->color_info.r);
		float diffG = fabs(si1->color_info.g - si2->color_info.g);
		float diffB = fabs(si1->color_info.b - si2->color_info.b);
		float colorDiff = sqrtf(diffR * diffR + diffG * diffG + diffB * diffB);
		return colorDiff;
	}

	void VClusterEngine::classifyObject(SegGraph *G, int i, int minPts) {
		if (G->nodes[i].numNeighbors >= minPts)
			G->nodes[i].type = NODE_CORE;
		else G->nodes[i].type = NODE_BORDER;
	}

	void VClusterEngine::makeGraph(int minPts, float Rs, float Rc, SPixelInfo *si, int numVertices, SegGraph *G) {
		G->nodes = new SegNode[numVertices];
		G->numV = numVertices;

		const dim3 block(10, 1);
		const dim3 grid(G->numV / block.x, 1);

		thrust::device_vector<int> Va1(numVertices);
		thrust::device_vector<int> Va2(numVertices);
		SPixelInfo *siPtr;
		SegNode *NPtr;

		ORcudaSafeCall(hipMalloc<struct SegNode>(&NPtr, sizeof(SegNode) * G->numV));
		ORcudaSafeCall(hipMalloc<struct SPixelInfo>(&siPtr, sizeof(SPixelInfo) * G->numV));
		ORcudaSafeCall(hipMemcpy(siPtr, si, sizeof(SPixelInfo) * G->numV, hipMemcpyHostToDevice));
		ORcudaSafeCall(hipMemcpy(NPtr, G->nodes, sizeof(SegNode) * G->numV, hipMemcpyHostToDevice));

		int* Va1Ptr = thrust::raw_pointer_cast(Va1.data());
		int* Va2Ptr = thrust::raw_pointer_cast(Va2.data());
		makeGraph_device_step1 << <grid, block >> >(siPtr, NPtr, Va1Ptr, numVertices, Rs, Rc);

		hipDeviceSynchronize();
		hipDeviceSynchronize();
		ORcudaSafeCall(hipMemcpy(G->nodes, NPtr, sizeof(SegNode)* G->numV, hipMemcpyDeviceToHost));

		thrust::exclusive_scan(Va1.begin(), Va1.end(), Va2.begin());

		G->numE = Va1[G->numV - 1] + Va2[G->numV - 1];
		G->Ea = new int[G->numE];
		ORcudaSafeCall(hipMalloc<int>(&G->e, sizeof(int)* G->numE));
		makeGraph_device_step2 << <grid, block >> >(siPtr, NPtr, Va1Ptr, Va2Ptr, G->e, numVertices, Rs, Rc);

		hipDeviceSynchronize();
		hipDeviceSynchronize();
		G->va2.clear();
		G->va2.reserve(G->numV);
		thrust::copy(Va2.begin(), Va2.end(), G->va2.begin());

		ORcudaSafeCall(hipMemcpy(G->Ea, G->e, sizeof(int)* G->numE, hipMemcpyDeviceToHost));

		for (int i = 0; i < G->numV; ++i) {
			classifyObject(G, i, minPts);
			G->nodes[i].ePtr = Va2[i];
		}

		ORcudaSafeCall(hipFree(siPtr));
		ORcudaSafeCall(hipFree(NPtr));
	}

	__global__ void makeGraph_device_step1(SPixelInfo* si, SegNode *nodes, int *Va1, int numV, float Rs, float Rc) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		SPixelInfo *thisSP = &si[tid];
		SegNode *thisNode = &nodes[tid];

		thisNode->type = NODE_BORDER;
		thisNode->numNeighbors = 0;
		thisNode->visited = false;
		thisNode->ePtr = -1;
		thisNode->cluster = -1;

		Va1[tid] = 0;

		for (int i = 0; i < numV; ++i) {
			float distS = distanceS(thisSP, &si[i]);
			float distC = distanceC(thisSP, &si[i]);
			if (tid != i && distS <= Rs && distC <= Rc) {
				thisNode->numNeighbors++;
			}
		}
		Va1[tid] = thisNode->numNeighbors;
	}

	__global__ void makeGraph_device_step2(SPixelInfo* si, SegNode *nodes, int *Va1, int *Va2, int *e, int numV, float Rs, float Rc) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		SPixelInfo *thisSP = &si[tid];

		int ePtr = Va2[tid];
		for (int i = 0; i < numV; ++i) {
			float distS = distanceS(thisSP, &si[i]);
			float distC = distanceC(thisSP, &si[i]);
			if (tid != i && distS <= Rs && distC <= Rc) {
				e[ePtr++] = i;
			}
		}
	}

	void VClusterEngine::identifyCluster(SegGraph *G) {
		int clusterID = 0;

		for (int i = 0; i < G->numV; ++i) {
			if (!G->nodes[i].visited && G->nodes[i].type == NODE_CORE) {
				G->nodes[i].visited = true;
				G->nodes[i].cluster = clusterID;
				G->clusterMap.insert(std::make_pair(clusterID, i));
				breadthFirstSearch(G, i, clusterID);
				clusterID++;
			}
		}

		G->numClusters = clusterID;
	}

	void VClusterEngine::breadthFirstSearch(SegGraph* G, int v, int clusterID) {
		int inputBytes = sizeof(unsigned char)* G->numV;

		unsigned char *XaPtr;
		unsigned char *FaPtr;
		SegNode *NPtr;
		ORcudaSafeCall(hipMalloc<unsigned char>(&XaPtr, inputBytes));
		ORcudaSafeCall(hipMalloc<unsigned char>(&FaPtr, inputBytes));
		ORcudaSafeCall(hipMalloc<struct SegNode>(&NPtr, sizeof(SegNode)* G->numV));
		ORcudaSafeCall(hipMemcpy(NPtr, G->nodes, sizeof(SegNode)* G->numV, hipMemcpyHostToDevice));

		unsigned char *Xa = new unsigned char[G->numV];
		unsigned char *Fa = new unsigned char[G->numV];
		memset(Xa, 0, sizeof(unsigned char)* G->numV);
		memset(Fa, 0, sizeof(unsigned char)* G->numV);
		Fa[v] = 1;
		ORcudaSafeCall(hipMemcpy(FaPtr, Fa, sizeof(unsigned char)* G->numV, hipMemcpyHostToDevice));
		ORcudaSafeCall(hipMemcpy(XaPtr, Xa, sizeof(unsigned char)* G->numV, hipMemcpyHostToDevice));
		int countFa = 1;

		const dim3 block(10, 1);
		const dim3 grid(G->numV / block.x, 1);

		int countLoop = 0;
		while (countFa > 0) {
			breadthFirstSearch_device << <grid, block >> >(NPtr, G->numV, G->e, FaPtr, XaPtr);
			hipDeviceSynchronize();
			hipDeviceSynchronize();
			ORcudaSafeCall(hipMemcpy(Fa, FaPtr, inputBytes, hipMemcpyDeviceToHost));

			countFa = thrust::count(thrust::device, FaPtr, FaPtr + G->numV, 1);
			countLoop++;
		}

		ORcudaSafeCall(hipMemcpy(Xa, XaPtr, inputBytes, hipMemcpyDeviceToHost));

		for (int i = 0; i < G->numV; ++i) {
			if (Xa[i]) {
				G->clusterMap.insert(std::make_pair(clusterID, i));
				G->nodes[i].cluster = clusterID;
				G->nodes[i].visited = true;
				if (G->nodes[i].type != NODE_CORE) {
					G->nodes[i].type = NODE_BORDER;
				}
			}
		}

		ORcudaSafeCall(hipFree(XaPtr));
		ORcudaSafeCall(hipFree(FaPtr));
		ORcudaSafeCall(hipFree(NPtr));
		delete[] Xa;
		delete[] Fa;
	}

	__global__ void breadthFirstSearch_device(SegNode *N, int numV, int *EPtr, unsigned char *Fa, unsigned char *Xa) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if (Fa[tid]) {
			Fa[tid] = 0;
			Xa[tid] = 1;

			int ePtr = N[tid].ePtr;
			for (int i = 0; i < N[tid].numNeighbors; ++i) {
				int nid = EPtr[ePtr + i];
				Fa[nid] = 1 - Xa[nid];
			}
		}
	}
}

