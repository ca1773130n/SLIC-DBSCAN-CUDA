#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of gSLICr

#include <sdbscan/VSegEngineGPU.h>
#include <sdbscan/VSegEngineShared.h>
#include <iostream>
#include <thrust/device_vector.h>

namespace sdbscan {
	// ----------------------------------------------------
	//
	//	kernel function defines
	//
	// ----------------------------------------------------

	__global__ void ConvertImgSpace_device(const Vector4u* inimg, Vector4f* outimg, Vector2i mImageSize, COLOR_SPACE mColorSpace);

	__global__ void EnforceConnectivity_device(const int* in_mIndexImg, int* out_mIndexImg, Vector2i mImageSize);

	__global__ void InitClusterCenters_device(const int* mSegColors, const Vector4f* inimg, SPixelInfo* accum_map, SPixelInfo* out_spixel, Vector2i map_size, Vector2i mImageSize, int mSpixelSize);

	__global__ void FindCenterAssociation_device(const Vector4f* inimg, const SPixelInfo* in_mSpixelMap, int* out_mIndexImg, Vector2i map_size, Vector2i mImageSize, int mSpixelSize, float weight, float mMaxPosDist, float mMaxColorDist);

	__global__ void UpdateClusterCenter_device(const Vector4f* inimg, const int* in_mIndexImg, SPixelInfo* accum_map, Vector2i map_size, Vector2i mImageSize, int mSpixelSize, int no_blocks_per_line);

	__global__ void FinalizeReductionResult_device(const SPixelInfo* accum_map, SPixelInfo* spixel_list, Vector2i map_size, int no_blocks_per_spixel);

	__global__ void DrawSegmentationResult_device(const SPixelInfo* in_mSpixelMap, const int mSpixelSize, const int* mIndexImg, Vector4u* orgimg, Vector4u* sourceimg, Vector4u* outimg, Vector4u* clusterimg, Vector4u* avgcolorimg, Vector4u* numpixelsimg, Vector4u* centerimg, Vector4u** ptrs, Vector2i mImageSize);

	// ----------------------------------------------------
	//
	//	host function implementations
	//
	// ----------------------------------------------------

	VSegEngineGPU::VSegEngineGPU(const VSettings& inputSettings) : VSegEngine(inputSettings)
	{
		mSourceImg = new UChar4Image(inputSettings.mImageSize, true, true);
		mConvertedImg = new Float4Image(inputSettings.mImageSize, true, true);
		mIndexImg = new IntImage(inputSettings.mImageSize, true, true);
		mTempIndexImg = new IntImage(inputSettings.mImageSize, true, true);
		ORcudaSafeCall(hipMalloc<int>(&mSegColors, sizeof(int)* inputSettings.mNumSegments));
		ORcudaSafeCall(hipMemcpy(mSegColors, inputSettings.mSegColors, inputSettings.mNumSegments * sizeof(int), hipMemcpyHostToDevice));
		if (inputSettings.mSegmentationMethod == GIVEN_NUM)
		{
			float cluster_size = (float)(inputSettings.mImageSize.x * inputSettings.mImageSize.y) / (float)inputSettings.mNumSegments;
			mSpixelSize = (int)ceil(sqrtf(cluster_size));
			inputSettings.mResImages->mSpixelImgs = new UChar4ImageArray(inputSettings.mNumSegments, Vector2i(mSpixelSize * 3, mSpixelSize * 3), true, true);
		}
		else
		{
			mSpixelSize = inputSettings.mSpixelSize;
		}

		int spixel_per_col = (int)ceil(inputSettings.mImageSize.x / mSpixelSize);
		int spixel_per_row = (int)ceil(inputSettings.mImageSize.y / mSpixelSize);

		Vector2i map_size = Vector2i(spixel_per_col, spixel_per_row);
		mSpixelMap = new SPixelMap(map_size, true, true);
		inputSettings.mResImages->mSpixelMap = mSpixelMap;

		float total_pixel_to_search = (float)(mSpixelSize * mSpixelSize * 9);
		no_grid_per_center = (int)ceil(total_pixel_to_search / (float)(BLOCK_DIM * BLOCK_DIM));

		map_size.x *= no_grid_per_center;
		accum_map = new ORUtils::Image<SPixelInfo>(map_size, true, true);

		// normalizing factors
		mMaxPosDist = 1.0f / (1.4242f * mSpixelSize); // sqrt(2) * mSpixelSize
		switch (inputSettings.mColorSpace)
		{
			case RGB:
				mMaxColorDist = 5.0f / (1.7321f * 255);
				break;
			case XYZ:
				mMaxColorDist = 5.0f / 1.7321f;
				break;
			case CIELAB:
				mMaxColorDist = 15.0f / (1.7321f * 128);
				break;
		}

		mMaxColorDist *= mMaxColorDist;
		mMaxPosDist *= mMaxPosDist;
	}

	VSegEngineGPU::~VSegEngineGPU()
	{
		delete accum_map;
		ORcudaSafeCall(hipFree(mSegColors));
	}


	void VSegEngineGPU::convertImgSpace(UChar4Image* inimg, Float4Image* outimg, COLOR_SPACE mColorSpace)
	{
		Vector4u* inimg_ptr = inimg->GetData(MEMORYDEVICE_CUDA);
		Vector4f* outimg_ptr = outimg->GetData(MEMORYDEVICE_CUDA);
		Vector2i mImageSize = inimg->noDims;

		dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
		dim3 gridSize((int)ceil((float)mImageSize.x / (float)blockSize.x), (int)ceil((float)mImageSize.y / (float)blockSize.y));

		ConvertImgSpace_device << <gridSize, blockSize >> >(inimg_ptr, outimg_ptr, mImageSize, mColorSpace);

	}

	void VSegEngineGPU::initClusterCenters()
	{
		SPixelInfo* accum_map_ptr = accum_map->GetData(MEMORYDEVICE_CUDA);
		SPixelInfo* spixel_list = mSpixelMap->GetData(MEMORYDEVICE_CUDA);
		Vector4f* img_ptr = mConvertedImg->GetData(MEMORYDEVICE_CUDA);

		Vector2i map_size = mSpixelMap->noDims;
		Vector2i mImageSize = mConvertedImg->noDims;

		dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
		dim3 gridSize((int)ceil((float)map_size.x / (float)blockSize.x), (int)ceil((float)map_size.y / (float)blockSize.y));

		InitClusterCenters_device << <gridSize, blockSize >> >(mSegColors, img_ptr, accum_map_ptr, spixel_list, map_size, mImageSize, mSpixelSize);
	}

	void VSegEngineGPU::findCenterAssociation()
	{
		SPixelInfo* spixel_list = mSpixelMap->GetData(MEMORYDEVICE_CUDA);
		Vector4f* img_ptr = mConvertedImg->GetData(MEMORYDEVICE_CUDA);
		int* idx_ptr = mIndexImg->GetData(MEMORYDEVICE_CUDA);

		Vector2i map_size = mSpixelMap->noDims;
		Vector2i mImageSize = mConvertedImg->noDims;

		dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
		dim3 gridSize((int)ceil((float)mImageSize.x / (float)blockSize.x), (int)ceil((float)mImageSize.y / (float)blockSize.y));

		FindCenterAssociation_device << <gridSize, blockSize >> >(img_ptr, spixel_list, idx_ptr, map_size, mImageSize, mSpixelSize, mSettings.mCohWeight, mMaxPosDist, mMaxColorDist);
	}

	void VSegEngineGPU::updateClusterCenter()
	{
		SPixelInfo* accum_map_ptr = accum_map->GetData(MEMORYDEVICE_CUDA);
		SPixelInfo* spixel_list_ptr = mSpixelMap->GetData(MEMORYDEVICE_CUDA);
		Vector4f* img_ptr = mConvertedImg->GetData(MEMORYDEVICE_CUDA);
		int* idx_ptr = mIndexImg->GetData(MEMORYDEVICE_CUDA);

		Vector2i map_size = mSpixelMap->noDims;
		Vector2i mImageSize = mConvertedImg->noDims;

		int no_blocks_per_line = mSpixelSize * 3 / BLOCK_DIM;

		dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
		dim3 gridSize(map_size.x, map_size.y, no_grid_per_center);

		UpdateClusterCenter_device << <gridSize, blockSize >> >(img_ptr, idx_ptr, accum_map_ptr, map_size, mImageSize, mSpixelSize, no_blocks_per_line);

		dim3 gridSize2(map_size.x, map_size.y);

		FinalizeReductionResult_device << <gridSize2, blockSize >> >(accum_map_ptr, spixel_list_ptr, map_size, no_grid_per_center);
		mSpixelMap->updateHostFromDevice();
	}

	void VSegEngineGPU::enforceConnectivity()
	{
		int* idx_ptr = mIndexImg->GetData(MEMORYDEVICE_CUDA);
		int* tmp_idx_ptr = mTempIndexImg->GetData(MEMORYDEVICE_CUDA);
		Vector2i mImageSize = mIndexImg->noDims;

		dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
		dim3 gridSize((int)ceil((float)mImageSize.x / (float)blockSize.x), (int)ceil((float)mImageSize.y / (float)blockSize.y));

		EnforceConnectivity_device << <gridSize, blockSize >> >(idx_ptr, tmp_idx_ptr, mImageSize);
		EnforceConnectivity_device << <gridSize, blockSize >> >(tmp_idx_ptr, idx_ptr, mImageSize);
	}

	void VSegEngineGPU::drawSegmentationResult(const SegResult *res)
	{
		SPixelInfo* sPixelListPtr = mSpixelMap->GetData(MEMORYDEVICE_CUDA);
		Vector4u* orgImgPtr = res->mOriginalImg->GetData(MEMORYDEVICE_CUDA);
		Vector4u* inImgPtr = mSourceImg->GetData(MEMORYDEVICE_CUDA);
		Vector4u* outImgPtr = res->mOutputImg->GetData(MEMORYDEVICE_CUDA);
		Vector4u* clusterImgPtr = res->mClusterImg->GetData(MEMORYDEVICE_CUDA);
		Vector4u* avgColorImgPtr = res->mAvgColorImg->GetData(MEMORYDEVICE_CUDA);
		Vector4u* numPixelsImgPtr = res->mNumPixelsImg->GetData(MEMORYDEVICE_CUDA);
		Vector4u* centerImgPtr = res->mCenterImg->GetData(MEMORYDEVICE_CUDA);
		int* idxImgPtr = mIndexImg->GetData(MEMORYDEVICE_CUDA);

		Vector2i mImageSize = mIndexImg->noDims;

		dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
		dim3 gridSize((int)ceil((float)mImageSize.x / (float)blockSize.x), (int)ceil((float)mImageSize.y / (float)blockSize.y));
		Vector4u** spixels = thrust::raw_pointer_cast(&res->mSpixelImgs->getPtrs()[0]);

		for (size_t i = 0; i < res->mNumSegments; ++i)
			res->mSpixelImgs->getImage(i)->Clear();

		DrawSegmentationResult_device << <gridSize, blockSize >> >(sPixelListPtr, res->mSpixelSize, idxImgPtr, orgImgPtr, inImgPtr, outImgPtr, clusterImgPtr, avgColorImgPtr, numPixelsImgPtr, centerImgPtr, spixels, mImageSize);

		hipDeviceSynchronize();

		res->mOutputImg->updateHostFromDevice();
		res->mClusterImg->updateHostFromDevice();
		res->mAvgColorImg->updateHostFromDevice();
		res->mNumPixelsImg->updateHostFromDevice();
		res->mCenterImg->updateHostFromDevice();
		for (size_t i = 0; i < res->mNumSegments; ++i)
			res->mSpixelImgs->getImage(i)->updateHostFromDevice();
	}

	// ----------------------------------------------------
	//
	//	device function implementations
	//
	// ----------------------------------------------------

	__global__ void ConvertImgSpace_device(const Vector4u* inimg, Vector4f* outimg, Vector2i mImageSize, COLOR_SPACE mColorSpace)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
		if (x > mImageSize.x - 1 || y > mImageSize.y - 1) return;

		ConvertImgSpace_shared(inimg, outimg, mImageSize, x, y, mColorSpace);

	}

	__global__ void DrawSegmentationResult_device(const SPixelInfo* in_mSpixelMap, const int mSpixelSize, const int* mIndexImg, Vector4u* orgimg, Vector4u* sourceimg, Vector4u* outimg, Vector4u* clusterimg, Vector4u* avgcolorimg, Vector4u* numpixelsimg, Vector4u* centerimg, Vector4u** spimgs, Vector2i mImageSize)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
		if (x == 0 || y == 0 || x > mImageSize.x - 2 || y > mImageSize.y - 2) return;

		DrawSuperPixelBoundry_shared(in_mSpixelMap, mSpixelSize, mIndexImg, orgimg, sourceimg, outimg, clusterimg, avgcolorimg, numpixelsimg, centerimg, spimgs, mImageSize, x, y);
	}

	__global__ void InitClusterCenters_device(const int* mSegColors, const Vector4f* inimg, SPixelInfo* accum_map, SPixelInfo* out_spixel, Vector2i map_size, Vector2i mImageSize, int mSpixelSize)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
		int g = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x)*blockDim.x);
		if (x > map_size.x - 1 || y > map_size.y - 1) return;

		InitClusterCenters_shared(mSegColors, inimg, accum_map, out_spixel, map_size, mImageSize, mSpixelSize, x, y, g);
	}

	__global__ void FindCenterAssociation_device(const Vector4f* inimg, const SPixelInfo* in_mSpixelMap, int* out_mIndexImg, Vector2i map_size, Vector2i mImageSize, int mSpixelSize, float weight, float mMaxPosDist, float mMaxColorDist)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
		if (x > mImageSize.x - 1 || y > mImageSize.y - 1) return;

		FindCenterAssociation_shared(inimg, in_mSpixelMap, out_mIndexImg, map_size, mImageSize, mSpixelSize, weight, x, y, blockDim.x, blockDim.y, mMaxPosDist, mMaxColorDist);
	}

	__global__ void UpdateClusterCenter_device(const Vector4f* inimg, const int* in_mIndexImg, SPixelInfo* accum_map, Vector2i map_size, Vector2i mImageSize, int mSpixelSize, int no_blocks_per_line)
	{
		int local_id = threadIdx.y * blockDim.x + threadIdx.x;

		__shared__ Vector4f color_shared[BLOCK_DIM*BLOCK_DIM];
		__shared__ Vector2f xy_shared[BLOCK_DIM*BLOCK_DIM];
		__shared__ int count_shared[BLOCK_DIM*BLOCK_DIM];
		__shared__ bool should_add;

		color_shared[local_id] = Vector4f(0, 0, 0, 0);
		xy_shared[local_id] = Vector2f(0, 0);
		count_shared[local_id] = 0;
		should_add = false;
		__syncthreads();

		int no_blocks_per_spixel = gridDim.z;

		int spixel_id = blockIdx.y * map_size.x + blockIdx.x;

		// compute the relative position in the search window
		int block_x = blockIdx.z % no_blocks_per_line;
		int block_y = blockIdx.z / no_blocks_per_line;

		int x_offset = block_x * BLOCK_DIM + threadIdx.x;
		int y_offset = block_y * BLOCK_DIM + threadIdx.y;

		if (x_offset < mSpixelSize * 3 && y_offset < mSpixelSize * 3)
		{
			// compute the start of the search window
			int x_start = blockIdx.x * mSpixelSize - mSpixelSize;
			int y_start = blockIdx.y * mSpixelSize - mSpixelSize;

			int x_img = x_start + x_offset;
			int y_img = y_start + y_offset;

			if (x_img >= 0 && x_img < mImageSize.x && y_img >= 0 && y_img < mImageSize.y)
			{
				int img_idx = y_img * mImageSize.x + x_img;
				if (in_mIndexImg[img_idx] == spixel_id)
				{
					int accum_map_idx = spixel_id * no_blocks_per_spixel + blockIdx.z;
					color_shared[local_id] = inimg[img_idx];
					xy_shared[local_id] = Vector2f(x_img, y_img);
					count_shared[local_id] = 1;
					should_add = true;
				}
			}
		}
		__syncthreads();

		if (should_add)
		{
			if (local_id < 128)
			{
				color_shared[local_id] += color_shared[local_id + 128];
				xy_shared[local_id] += xy_shared[local_id + 128];
				count_shared[local_id] += count_shared[local_id + 128];
			}
			__syncthreads();

			if (local_id < 64)
			{
				color_shared[local_id] += color_shared[local_id + 64];
				xy_shared[local_id] += xy_shared[local_id + 64];
				count_shared[local_id] += count_shared[local_id + 64];
			}
			__syncthreads();

			if (local_id < 32)
			{
				color_shared[local_id] += color_shared[local_id + 32];
				color_shared[local_id] += color_shared[local_id + 16];
				color_shared[local_id] += color_shared[local_id + 8];
				color_shared[local_id] += color_shared[local_id + 4];
				color_shared[local_id] += color_shared[local_id + 2];
				color_shared[local_id] += color_shared[local_id + 1];

				xy_shared[local_id] += xy_shared[local_id + 32];
				xy_shared[local_id] += xy_shared[local_id + 16];
				xy_shared[local_id] += xy_shared[local_id + 8];
				xy_shared[local_id] += xy_shared[local_id + 4];
				xy_shared[local_id] += xy_shared[local_id + 2];
				xy_shared[local_id] += xy_shared[local_id + 1];

				count_shared[local_id] += count_shared[local_id + 32];
				count_shared[local_id] += count_shared[local_id + 16];
				count_shared[local_id] += count_shared[local_id + 8];
				count_shared[local_id] += count_shared[local_id + 4];
				count_shared[local_id] += count_shared[local_id + 2];
				count_shared[local_id] += count_shared[local_id + 1];
			}
		}
		__syncthreads();

		if (local_id == 0)
		{
			int accum_map_idx = spixel_id * no_blocks_per_spixel + blockIdx.z;
			accum_map[accum_map_idx].center = xy_shared[0];
			accum_map[accum_map_idx].color_info = color_shared[0];
			accum_map[accum_map_idx].no_pixels = count_shared[0];
		}
	}

	__global__ void FinalizeReductionResult_device(const SPixelInfo* accum_map, SPixelInfo* spixel_list, Vector2i map_size, int no_blocks_per_spixel)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
		if (x > map_size.x - 1 || y > map_size.y - 1) return;

		FinalizeReductionResult_shared(accum_map, spixel_list, map_size, no_blocks_per_spixel, x, y);
	}

	__global__ void EnforceConnectivity_device(const int* in_mIndexImg, int* out_mIndexImg, Vector2i mImageSize)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
		if (x > mImageSize.x - 1 || y > mImageSize.y - 1) return;

		SupressLocalLable(in_mIndexImg, out_mIndexImg, mImageSize, x, y);
	}
}

